#include "hip/hip_runtime.h"
/* 
 * File:   MBSet.cu
 * 
 * Created on June 24, 2012, edited 12/07/14 by Tom Wells as part of ECE 4993 project work.
 * 
 * Purpose:  This program displays Mandelbrot set using the GPU via CUDA and
 * OpenGL immediate mode.
 * 
 */

#include <iostream>
#include <stack>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "Complex.cu"

#include <GL/freeglut.h>
//#include <GL/gl.h>
//#include <GL/glu.h>
//#include <GL/glut.h>
#define WINDOW_DIM            512
#define THREADS		       32
#define BLOCKS		     8192
#define THRESH_SQ		4
#define MAXIT		     2000

__global__ void getColor(int *colorIndex, Complex* min, Complex* max )
{
	int count = 0;
	float dr = (max->r - min->r) / WINDOW_DIM;
	float di = (max->i - min->i) / WINDOW_DIM;
	//mapping
	float real = (blockIdx.x %(WINDOW_DIM / THREADS) + threadIdx.x) * dr;
	float img  = (blockIdx.x / (WINDOW_DIM / THREADS))  *di;
	Complex Z_0(real, img);
	Complex Z_I(Z_0); 
	//itterate Z_n = Z_n-1 ^2 + Z_0
	while(count <=MAXIT && Z_I.magnitude2() <=THRESH_SQ)//magnitude^2 <= 4 iff magnitude <=2
	{
		count++;
		Z_I = (Z_I * Z_I) + Z_0;
	}
	colorIndex[(blockIdx.x * THREADS) + threadIdx.x] = count;
}
using namespace std;

// Initial screen coordinates, both host and device.
// Define the RGB Class
class RGB
{
public:
  RGB()
    : r(0), g(0), b(0) {}
  RGB(double r0, double g0, double b0)
    : r(r0), g(g0), b(b0) {}
public:
  double r;
  double g;
  double b;
};

RGB* colors = 0; // Array of color values
Complex minC(-2.0, -1.2);
Complex maxC(1.0, 1.8);

Complex zoomFactorR(.9,0);
Complex zoomFactorI(0,.9);
int zoomCount = 0;

int *colorIndex;
void InitializeColors()
{
  colors = new RGB[MAXIT + 1];
  for (int i = 0; i < MAXIT; ++i)
    {
      if (i < 5)
        { // Try this.. just white for small it counts
          colors[i] = RGB(1, 1, 1);
        }
      else
        {
          colors[i] = RGB(drand48(), drand48(), drand48());
        }
    }
  colors[MAXIT] = RGB(); // black
}
void keyboard(unsigned char key, int x, int y)
{
	if(key=='q')
	{
		exit(0);
	}
}
void mouse(int button, int state, int x, int y)
{
	//onclick increment zoom count
	//change center point
}
void display(void)
{
//ttrrwsrt
	glClearColor(0.0,0.0,0.0,1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0, 1, 1, 0, -1, 1);
	//loop through vertex2d of colors defined in color index
//	for(int i = 0; i<zoomCount; i++)
//	{
//		minC = minC * zoomFactorR;
//		minC = minC * zoomFactorI;

//		maxC = maxC * zoomFactorR;
//		maxC = maxC * zoomFactorI;
//	}
	glBegin(GL_POINTS);
	for(int i = 0; i<WINDOW_DIM; i++)
	{
		for(int j = 0; j<WINDOW_DIM; i++)
		{
			RGB tempC = colors[colorIndex[i*WINDOW_DIM + j]];
			glColor3f(tempC.r, tempC.g, tempC.b);
			glVertex2f((1.0* i)/WINDOW_DIM, (1.0 * j)/WINDOW_DIM);
			
		}
	}
	glEnd();
	glutSwapBuffers();
}
int main(int argc, char** argv)
{
  // Initialize OPENGL here
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowPosition(100,100);
	glutInitWindowSize(WINDOW_DIM, WINDOW_DIM);
	glViewport(0,0,(GLsizei)WINDOW_DIM, (GLsizei)WINDOW_DIM);
	glEnable(GL_DEPTH_TEST);

	
	glutCreateWindow("Mandelbrot Set");
  // Set up necessary host and device buffers
  // set up the opengl callbacks for display, mouse and keyboard

  // Calculate the interation counts
  // Grad students, pick the colors for the 0 .. 1999 iteration count pixels
	int size = WINDOW_DIM * WINDOW_DIM;//total number of pixels
	InitializeColors();
	//host buffers
	colorIndex = (int*)malloc(size);
	//device buffers
	int *d_colorIndex;
	hipMalloc((void **)&d_colorIndex, size);
	//window converions bounds

	Complex* d_minC;
	Complex* d_maxC;
	hipMalloc((void **)&d_minC, sizeof(Complex));
	hipMalloc((void **)&d_maxC, sizeof(Complex)); 

	glutDisplayFunc(display);
	glutIdleFunc(display);
//	glutKeyboardFunc(keyboard);
//	glutMouseFunc(mouse);

	//move values to cuda
	hipMemcpy(d_minC, &minC, sizeof(Complex), hipMemcpyHostToDevice);
	hipMemcpy(d_maxC, &maxC, sizeof(Complex), hipMemcpyHostToDevice);

	getColor<<<(WINDOW_DIM*WINDOW_DIM) /THREADS, THREADS>>>(d_colorIndex, d_minC, d_maxC);

	hipMemcpy(colorIndex, d_colorIndex, size, hipMemcpyDeviceToHost);
	for(int i = 0; i<512; i++)
	{
		cout<<colorIndex[i]<<"\n";
	} 
//	glutMainLoop(); // THis will callback the display, keyboard and mouse
	free(colorIndex);
	hipFree(d_colorIndex);
	hipFree(d_minC);
	hipFree(d_maxC);
	return 0;
  
}
